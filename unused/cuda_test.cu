
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to add two vectors
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
        printf("%d\t", c[idx]);
    }
}

int main() {
    const int N = 256;  // Size of vectors

    // Host vectors
    int *h_a = (int *)malloc(N * sizeof(int));
    int *h_b = (int *)malloc(N * sizeof(int));
    int *h_c = (int *)malloc(N * sizeof(int));

    // Initialize host vectors
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    // Device vectors
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, N * sizeof(int));
    hipMalloc((void **)&d_b, N * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));

    // Transfer data from host to device
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 16;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Transfer results from device to host
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < N; ++i) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("Error: mismatch at index %d, expected %d, got %d\n", i, h_a[i] + h_b[i], h_c[i]);
        }
    }

    printf("Vector addition on GPU successfully completed!\n");

    // Clean up
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
